/*#include <boost/random.hpp>
#include <cstdlib>
#include <cstdio>
#include <vector>
#include <chrono>
#include <algorithm>
#include <numeric>

#define N 5000

using namespace std ; */


#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
  printf("No device :(") ; 
}


/*
int main (int argc, char *argv[])
{
 boost::random::mt19937 rng; 
 boost::random::uniform_01<boost::mt19937> rand(rng) ; 
 auto start = std::chrono::high_resolution_clock::now();
 auto elapsed = std::chrono::high_resolution_clock::now()-start;   
 
 vector <vector <double> > particles (N, vector<double>(3,0))  ; 
 for (auto & v : particles)
     for (auto & w: v)
         w = rand() ; 
 
 double Dsqr=0.05 ; 
 
 start = std::chrono::high_resolution_clock::now();
 double sum ; 
 int count=0 ; 
 for (int i=0 ; i<N ; i++)
     for (int j=i+1 ; j<N ; j++)
     {
      sum=0 ; 
      for (int k=0 ; k<3 ; k++)
          sum += (particles[i][k]-particles[j][k])*(particles[i][k]-particles[j][k]) ; 
      if (sum < Dsqr)
          count ++ ;
     }
 elapsed = std::chrono::high_resolution_clock::now()-start; 
 auto duration= std::chrono::duration_cast<std::chrono::microseconds>(elapsed).count() ; 
 
 printf("%g %d\n", duration/1000000., count) ; 
    
}
*/
